#include "hip/hip_runtime.h"
#include "cutlass/half.h"
#include "flash_decoding/flash_decoding.h"
#include "flash_decoding/static_switch.h"
#include "tensor.h"

/**
 * @brief Set up parameters for multi-head attention (MHA) decoding forward pass.
 *
 * @param Q Tensor representing the query matrix.
 * @param K Tensor representing the key matrix.
 * @param V Tensor representing the value matrix.
 * @param O Tensor representing the output matrix.
 * @param cu_seqlens_q Tensor holding sequence lengths for queries.
 * @param cu_seqlens_k Tensor holding sequence lengths for keys.
 * @param seqlens_q Size of the query sequence length.
 * @param seqlens_k Size of the key sequence length.
 * @param stream CUDA stream for asynchronous execution.
 * @param dev_prop CUDA device properties.
 *
 * @return FlashDecodingParams struct filled with necessary parameters for the decoding process.
 */
FlashDecodingParams set_flash_decoding_params(Tensor<half> *Q, Tensor<half> *K, Tensor<half> *V, Tensor<half> *O,
                                              Tensor<int> *cu_seqlens_q, Tensor<int> *cu_seqlens_k, size_t seqlens_q,
                                              size_t seqlens_k, hipStream_t stream, hipDeviceProp_t *dev_prop)
{
    size_t num_q_head = Q->getShape()[1];
    size_t head_dim = Q->getShape()[2];
    size_t num_k_head = K->getShape()[1];
    size_t batch_size = cu_seqlens_q->getShape()[0] - 1;

    FAI_CHECK_LE(head_dim, 256);
    FAI_CHECK_EQ(num_q_head % num_k_head, 0);

    FlashDecodingParams params;

    // Reset the parameters
    memset(&params, 0, sizeof(params));

    // Set the pointers and strides.
    params.q_ptr = Q->getDevPtr();
    params.k_ptr = K->getDevPtr();
    params.v_ptr = V->getDevPtr();

    params.q_row_stride = num_q_head * head_dim;
    params.k_row_stride = num_k_head * head_dim;
    params.v_row_stride = num_k_head * head_dim;
    params.q_head_stride = head_dim;
    params.k_head_stride = head_dim;
    params.v_head_stride = head_dim;

    params.num_q_head = num_q_head;
    params.num_k_head = num_k_head;
    params.q_k_head_ratio = num_q_head / num_k_head;

    // Set the params for O = softmax(QK^T/sqrt(dk))V
    params.o_ptr = O->getDevPtr();
    params.o_row_stride = num_q_head * head_dim;
    params.o_head_stride = head_dim;

    // Set the dimensions
    params.batch_size = batch_size;
    params.seqlen_q = seqlens_q;
    params.seqlen_k = seqlens_k;
    params.head_dim = head_dim;

    params.scale_softmax = 1.0f / std::sqrt(static_cast<float>(head_dim));

    params.cu_seqlens_q = cu_seqlens_q->getDevPtr();
    params.cu_seqlens_k = cu_seqlens_k->getDevPtr();

    params.stream = stream;
    params.props = dev_prop;

    return params;
}

/**
 * @brief Execute the forward pass of multi-head attention decoding.
 *
 * @param params The FlashDecodingParams struct containing all necessary parameters for the forward pass.
 */
void run_flash_decoding_new(const FlashDecodingParams &params)
{
    DECODING_FWD_HEADDIM_SWITCH(params.head_dim, [&]
                                { run_flash_decoding_new_<kHeadDim>(params); });
}

/**
 * @brief High-level function to perform new decoding attention operation.
 *
 * @param Q Tensor representing the query matrix.
 * @param K Tensor representing the key matrix.
 * @param V Tensor representing the value matrix.
 * @param O Tensor representing the output matrix.
 * @param cu_seqlens_q Tensor holding sequence lengths for queries.
 * @param cu_seqlens_k Tensor holding sequence lengths for keys.
 * @param seqlens_q Size of the query sequence length.
 * @param seqlens_k Size of the key sequence length.
 * @param is_causal Boolean indicating if the operation is causal.
 * @param num_splits Number of splits in the operation.
 * @param stream CUDA stream for asynchronous execution.
 * @param dev_prop CUDA device properties.
 * @param is_alibi Boolean indicating if alibi is used.
 */
void flash_decoding(Tensor<half> *Q, Tensor<half> *K, Tensor<half> *V, Tensor<half> *O,
                    Tensor<int> *cu_seqlens_q, Tensor<int> *cu_seqlens_k, size_t seqlens_q, size_t seqlens_k,
                    bool is_causal, int num_splits, hipStream_t stream, hipDeviceProp_t *dev_prop, bool is_alibi)
{
    static FlashDecodingParams params = set_flash_decoding_params(Q, K, V, O, cu_seqlens_q, cu_seqlens_k, seqlens_q, seqlens_k, stream, dev_prop);

    FLOG("------------------Flash Decoding Params------------------");
    FLOG("seq_len_q: %d, seq_len_k: %d, batch_size: %d, head_dim: %d", params.seqlen_q, params.seqlen_k, params.batch_size, params.head_dim);
    FLOG("num_q_head: %d, num_k_head: %d, q_k_head_ratio: %d", params.num_q_head, params.num_k_head, params.q_k_head_ratio);
    FLOG("q_row_stride: %d, k_row_stride: %d, v_row_stride: %d, q_head_stride: %d, k_head_stride: %d, v_head_stride: %d", params.q_row_stride, params.k_row_stride, params.v_row_stride, params.q_head_stride, params.k_head_stride, params.v_head_stride);
    cu_seqlens_q->printTensor();
    cu_seqlens_k->printTensor();

    run_flash_decoding_new(params);
}
