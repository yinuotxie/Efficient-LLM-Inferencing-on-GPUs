#include "hip/hip_runtime.h"
// Copyright 2023. All Rights Reserved.
// Author: Bruce-Lee-LY
// Date: 21:14:13 on Tue, Oct 31, 2023
//
// Description: decoding attn

#include "decoding_attn/decoding.h"
#include "decoding_attn/static_switch.h"
#include "tensor.h"

DecodingParams set_mha_decoding_fwd_params(Tensor<half> *Q, Tensor<half> *K, Tensor<half> *V, Tensor<half> *O,
                                           Tensor<int> *cu_seq_q, Tensor<int> *cu_seq_k, size_t max_seq_q,
                                           size_t max_seq_k, hipStream_t stream, hipDeviceProp_t *dev_prop,
                                           bool is_alibi)
{
    size_t head_q = Q->getShape()[1];
    size_t dim = Q->getShape()[2];
    size_t head_k = K->getShape()[1];
    size_t batch = cu_seq_q->getShape()[0] - 1;

    FAI_CHECK_LE(dim, 256);
    FAI_CHECK_EQ(head_q % head_k, 0);

    DecodingParams params;

    // Reset the parameters
    memset(&params, 0, sizeof(params));

    // Set the pointers and strides.
    params.q_ptr = Q->getDevPtr();
    params.k_ptr = K->getDevPtr();
    params.v_ptr = V->getDevPtr();

    params.q_row_stride = head_q * dim; // used for going from one token to the next
    params.k_row_stride = head_k * dim;
    params.v_row_stride = head_k * dim;
    params.q_head_stride = dim; // used for going from one head to the next
    params.k_head_stride = dim;
    params.v_head_stride = dim;

    params.h = head_q;
    params.h_k = head_k;
    params.h_h_k_ratio = params.h / params.h_k;

    // O = softmax(QK^T / sqrt(d))V = [batch, head, seqlen, dim]
    params.o_ptr = O->getDevPtr();

    params.o_row_stride = head_q * dim;
    params.o_head_stride = dim;

    // Set the dimensions.
    params.b = batch;
    params.seqlen_q = max_seq_q;
    params.seqlen_k = max_seq_k;
    params.d = dim;

    params.scale_softmax = 1.0 / std::sqrt(dim);

    params.cu_seqlens_q = cu_seq_q->getDevPtr();
    params.cu_seqlens_k = cu_seq_k->getDevPtr();

    params.stream = stream;
    params.props = dev_prop;

    params.is_alibi = is_alibi;

    return params;
}

void run_mha_decoding_fwd(const DecodingParams &params)
{
    DECODING_FWD_HEADDIM_SWITCH(params.d, [&]
                                { run_mha_decoding_fwd_<HeadDim>(params); });
}

void decoding_attn(Tensor<half> *Q, Tensor<half> *K, Tensor<half> *V, Tensor<half> *O, Tensor<int> *cu_seq_q,
                   Tensor<int> *cu_seq_k, size_t max_seq_q, size_t max_seq_k, bool is_causal, int num_splits,
                   hipStream_t stream, hipDeviceProp_t *dev_prop, bool is_alibi)
{
    static DecodingParams params =
        set_mha_decoding_fwd_params(Q, K, V, O, cu_seq_q, cu_seq_k, max_seq_q, max_seq_k, stream, dev_prop, is_alibi);
    run_mha_decoding_fwd(params);
}
